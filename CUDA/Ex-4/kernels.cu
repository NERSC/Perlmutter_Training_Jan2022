#include "hip/hip_runtime.h"

#include <iostream>
#include "kernels.h"

#define ERRCHECK(ans)                                                                  \
{                                                                                    \
    gpuAssert((ans), __FILE__, __LINE__);                                            \
}
inline void
gpuAssert(hipError_t code, const char* file, int line, bool abort = true){
    if(code != hipSuccess){
        fprintf(stderr, "GPUassert: %s %s %d cpu:%d\n", hipGetErrorString(code), file, line);
        if(abort)
            exit(code);
    }
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}


int find_gpus(void) {
    int ngpu;
    hipGetDeviceCount(&ngpu);
    return ngpu;
}
void gpu_pci_id(char* device_id, int device_num) {
    int len=15;
    hipDeviceGetPCIBusId(device_id, len, device_num);
}

void vec_add_gpu(double *h_a, double *h_b, double *h_c, int n){
    // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;
    // Allocate memory for each vector on GPU
    size_t bytes = n*sizeof(double);
 
    ERRCHECK(hipMalloc(&d_a, bytes));
    ERRCHECK(hipMalloc(&d_b, bytes));
    ERRCHECK(hipMalloc(&d_c, bytes));

    // Copy host vectors to device
    ERRCHECK(hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice));
 
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 1024;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
 
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
 
    // Copy array back to host
    ERRCHECK(hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost ));

        // Release device memory
    ERRCHECK(hipFree(d_a));
    ERRCHECK(hipFree(d_b));
    ERRCHECK(hipFree(d_c));

}
