#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <mpi.h>

#define VECSIZE 100000


#define ERRCHECK(ans)                                                                  \
{                                                                                    \
    gpuAssert((ans), __FILE__, __LINE__);                                            \
}
inline void
gpuAssert(hipError_t code, const char* file, int line, bool abort = true){
    if(code != hipSuccess){
        fprintf(stderr, "GPUassert: %s %s %d cpu:%d\n", hipGetErrorString(code), file, line);
        if(abort)
            exit(code);
    }
}

int find_gpus(void) {
    int ngpu;
    hipGetDeviceCount(&ngpu);
    return ngpu;
}
void gpu_pci_id(char* device_id, int device_num){
    int len=15;
    hipDeviceGetPCIBusId(device_id, len, device_num);
}
void set_my_device(int my_device){
    ERRCHECK(hipSetDevice(my_device));
}
int get_current_device(){
    int my_device = -1;
    ERRCHECK(hipGetDevice(&my_device));

    return my_device;
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

void vec_add_gpu(double *h_a, double *h_b, double *h_c, int n){
    // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;
    // Allocate memory for each vector on GPU
    size_t bytes = n*sizeof(double);
 
    ERRCHECK(hipMalloc(&d_a, bytes));
    ERRCHECK(hipMalloc(&d_b, bytes));
    ERRCHECK(hipMalloc(&d_c, bytes));

    // Copy host vectors to device
    ERRCHECK(hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice));
    ERRCHECK(hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice));
 
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 1024;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
 
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
 
    // Copy array back to host
    ERRCHECK(hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost ));

        // Release device memory
    ERRCHECK(hipFree(d_a));
    ERRCHECK(hipFree(d_b));
    ERRCHECK(hipFree(d_c));

}

int main( int argc, char* argv[] )
{
    int myid, namelen, world_size;
    char myname[MPI_MAX_PROCESSOR_NAME];
    double final_result = 0;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &myid);
    MPI_Get_processor_name(myname, &namelen);
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    // fprintf(stdout, "Hello from processor %s, rank = %d out of %d processors" "\n", myname, myid, world_size);
    int ngpu = find_gpus();
    int my_gpu = myid%ngpu;
    char my_gpu_id[15];
    gpu_pci_id(my_gpu_id, my_gpu);
    fprintf(stdout, "Rank %d/%d from %s sees %d GPUs, GPU assigned to me is: = %s\n",myid, world_size, myname, ngpu, my_gpu_id);
    fprintf(stdout, "Other %d GPUs are: \n", (ngpu-1));

    for (int j = 0; j < ngpu; j++) {
    if (j != my_gpu) {
        char gpu_id[15];
        gpu_pci_id(gpu_id, j);
        fprintf(stdout, "**rank = %d: %s ** \n", j, gpu_id);
    }
    }

    //setting device for current GPU
    set_my_device(my_gpu);
    int curr_device = get_current_device();
    if(my_gpu != curr_device){
        fprintf(stderr, "********Device was not set properly for some ranks*******\n");
    }
    // Size of vectors
    int n = VECSIZE;
 
    // Host input vectors
    double *h_a;
    double *h_b;
    //Host output vector
    double *h_c;
 

    // Allocate memory for each vector on host
    h_a = new double [VECSIZE];//(double*)malloc(bytes);
    h_b = new double [VECSIZE];
    h_c = new double [VECSIZE];
 

    int i;
    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }
 
    vec_add_gpu(h_a, h_b, h_c, n);

    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    sum /=n;
    
    
    double global_sum = 0;

    MPI_Reduce(&sum, &global_sum, 1, MPI_DOUBLE, MPI_SUM, 0, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);

    // Release host memory
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    if(myid == 0)
        final_result = global_sum/(double)world_size;

    MPI_Finalize();
    
    if(myid == 0){
        if(final_result > 1.0){
            fprintf(stderr, "*****Result is incorrect, something went wrong, program will be terminated*****\n");
            exit(-1);
        }
        fprintf(stdout,"****final result: %f ******\n", final_result);
    }
    return 0;
}
